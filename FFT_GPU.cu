//**************************************************************
// Assignment 4
// Jose Soto and Nathan Van De Vyvere
// Parallel Programming Date: 11/03/2022
//**************************************************************
// Runs on maverick2
// sbatch a5Script
//
// Documentation: This program will compute FFT Radix-2
// with 8192 Samples. The host will create the sample
// table. The host calls the compute kernel where the device
// will divide up the discrete FFT to be computed.
//*************************************************************
//*************************************************************


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

const int SIZE = 8192;
const int BLOCKS = 4;
const int MAX_THREAD = 1024;
const double PI = 6.283185307179586;    // 2 * PI -- 15 digits after the decimal

//////////////////////////////////////////////////////////////////////////////////////////////////
/*
Function Name: compute
Parameters: double* k_vector_real, double* k_vector_imag, double* real_sample, double* imag_sample
Retunrn type: void
Description: This kernal will do the computations for each sample of the FFT. The kernal brings in
the sample data from the host. Each thread will calculate the even and odd parts for the k that 
matches its global index. The computed data is indexed into an array an passed back to the host.
*/
//////////////////////////////////////////////////////////////////////////////////////////////////

__global__
void compute(double* k_vector_real, double *k_vector_imag, double* real_sample, double* imag_sample)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;          // global index
  double COS_PC, SIN_PC, PC;                              // variables for the pi constant
  double EVEN_REAL, EVEN_IMAG, ODD_REAL, ODD_IMAG;        // variables for calculations

  // where sample data = x + yi
  // where PC is the pi constant for the even/odd summations
  // e^(-PCi) = cos(PC) - isin(PC)
  // even/odd summation = (x + yi) * [cos(PC) - isin(PC)]
  //                    = xcos(PC) - xisin(PC) + yicos(PC) + ysin(PC)
  //               real = xcos(PC) + ysin(PC)
  //                odd = yisin(PC) - xisin(PC)
  // summation loop for both the even and odd parts
  for (int m = 0; m < SIZE/2; m++)
  {
    PC = (PI * m * k)/(SIZE/2);
    COS_PC = cos(PC);
    SIN_PC = sin(PC);
    EVEN_REAL += real_sample[m+m] * COS_PC + imag_sample[m+m] * SIN_PC;
    ODD_REAL += real_sample[m+m+1] * COS_PC + imag_sample[m+m+1] * SIN_PC;
    EVEN_IMAG += imag_sample[m+m] * COS_PC - real_sample[m+m] * SIN_PC;
    ODD_IMAG += imag_sample[m+m+1] * COS_PC - real_sample[m+m+1] * SIN_PC;
  } 
  PC = PI * k / SIZE;   //twiddle factor
  // twiddle factor multiplied by the odd summation
  COS_PC = cos(PC);
  SIN_PC = sin(PC);
  ODD_REAL = ODD_REAL * COS_PC + ODD_IMAG * SIN_PC;
  ODD_IMAG = ODD_IMAG * COS_PC - ODD_REAL * SIN_PC;

  // for each k -> SUM(even) + twiddle factor * SUM(odd)
  k_vector_real[k] = EVEN_REAL + ODD_REAL;
  k_vector_imag[k] = EVEN_IMAG + ODD_IMAG;

  // for each k+N/2 -> SUM(even) - twiddle factor * SUM(odd)
  k_vector_real[k+(SIZE/2)] = EVEN_REAL - ODD_REAL;
  k_vector_imag[k+(SIZE/2)] = EVEN_IMAG - ODD_IMAG;
}

int main(void) {
  double k_vector_real[SIZE], k_vector_imag[SIZE];
  double real_sample[SIZE] = {3.6, 2.9, 5.6, 4.8, 3.3, 5.9, 5.0, 4.3};
  double imag_sample[SIZE] = {2.6, 6.3, 4.0, 9.1, 0.4, 4.8, 2.6, 4.1};

  double *real_sample_d, *imag_sample_d, *k_vector_real_d, *k_vector_imag_d;
  int size_d = SIZE * sizeof(double);

  // allocates memory in the GPU
  hipMalloc((void**) &real_sample_d, size_d);
  hipMalloc((void**) &imag_sample_d, size_d);
  hipMalloc((void**) &k_vector_real_d, size_d);
  hipMalloc((void**) &k_vector_imag_d, size_d);

  // copies the sample data to the allocated memory in the GPU
  hipMemcpy(real_sample_d, real_sample, size_d, hipMemcpyHostToDevice);
  hipMemcpy(imag_sample_d, imag_sample, size_d, hipMemcpyHostToDevice);
  
  // allocates the blocks in the GPU on the x-axis
  dim3 dimGrid(BLOCKS,1);
  // allocates the number of threads per block in the GPU
  dim3 dimBlock(MAX_THREAD,1);
  // calls the kernal function for computation
  compute <<< dimGrid, dimBlock >>> (k_vector_real_d, k_vector_imag_d, real_sample_d, imag_sample_d);
  
  // copies the vectors with the results from the GPU back to the host
  hipMemcpy(k_vector_real, k_vector_real_d, size_d, hipMemcpyDeviceToHost);
  hipMemcpy(k_vector_imag, k_vector_imag_d, size_d, hipMemcpyDeviceToHost);
  // cudaMemcpy(real_sample, real_sample_d, size_d, cudaMemcpyDeviceToHost);

  printf("\nTOTAL PROCESSED SAMPLES : %d\n",SIZE);
  printf("==========================================\n");
  for (int i = 0; i < 8; i++)
  {
    printf("XR[%d]: %.6f XI[%d]: %.6f \n", i, k_vector_real[i], i, k_vector_imag[i]);
    printf("==========================================\n");
  }
  printf("==========================================\n");
  for (int i = 4096; i < (4096+8); i++)
  {
    printf("XR[%d]: %.6f XI[%d]: %.6f \n", i, k_vector_real[i], i, k_vector_imag[i]);
    printf("==========================================\n");
  }

  // deallocates memory from the GPU
  hipFree(k_vector_imag_d);
  hipFree(k_vector_real_d);
  hipFree(real_sample_d);
  hipFree(imag_sample_d);

  return 0;
}
